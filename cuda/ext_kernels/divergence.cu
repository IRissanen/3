
#include <hip/hip_runtime.h>
// divergence of a field

extern "C" __global__ void
DivergenceCalc(float* __restrict__ FieldX, float* __restrict__ FieldY, float* __restrict__ FieldZ, float* __restrict__ divergence, int Nx, int Ny, int Nz, float* includeGeometry, float cellsize) {

	int ix = blockIdx.x * blockDim.x + threadIdx.x;
	int iy = blockIdx.y * blockDim.y + threadIdx.y;
	int iz = blockIdx.z * blockDim.z + threadIdx.z;

	if(ix >= Nx || iy >=  Ny || iz >= Nz) {
		return;
	}


	/*if(ix >= Nx-1 || iy >=  Ny-1 || iz >= Nz-1 || ix <= 0 || iy <= 0 || iz <= 0) {
		return;
	}*/

	int idx = ((iz)*(Ny) + (iy))*(Nx) + (ix);

	int pix = ((iz)*(Ny) + (iy))*(Nx) + ((ix-1+Nx)%Nx);
	int nix = ((iz)*(Ny) + (iy))*(Nx) + ((ix+1)%Nx);
	int piy = ((iz)*(Ny) + ((iy-1+Ny)%Ny))*(Nx) + (ix);
	int niy = ((iz)*(Ny) + ((iy+1)%Ny))*(Nx) + (ix);
	int piz = (((iz-1+Nz)%Nz)*(Ny) + (iy))*(Nx) + (ix);
	int niz = (((iz+1)%Nz)*(Ny) + (iy))*(Nx) + (ix);

	if(includeGeometry[idx] == 0 || includeGeometry[pix] == 0 || includeGeometry[nix] == 0 || includeGeometry[piy] == 0 || includeGeometry[niy] == 0 || includeGeometry[piz] == 0 || includeGeometry[niz] == 0)
	{
		return;
	}

	float px = FieldX[pix];
	float nx = FieldX[nix];
	float py = FieldY[piy];
	float ny = FieldY[niy];
	float pz = FieldZ[piz];
	float nz = FieldZ[niz];

	divergence[idx] = ((nx+ny+nz)-(px+py+pz))/(2.0*cellsize);

	}

